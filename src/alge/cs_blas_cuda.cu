/*============================================================================
 * Low-level functions and global variables definition for CUDA.
 *============================================================================*/

/*
  This file is part of Code_Saturne, a general-purpose CFD tool.

  Copyright (C) 1998-2022 EDF S.A.

  This program is free software; you can redistribute it and/or modify it under
  the terms of the GNU General Public License as published by the Free Software
  Foundation; either version 2 of the License, or (at your option) any later
  version.

  This program is distributed in the hope that it will be useful, but WITHOUT
  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
  FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more
  details.

  You should have received a copy of the GNU General Public License along with
  this program; if not, write to the Free Software Foundation, Inc., 51 Franklin
  Street, Fifth Floor, Boston, MA 02110-1301, USA.
*/

/*----------------------------------------------------------------------------*/

#include "cs_defs.h"

/*----------------------------------------------------------------------------
 * Standard C library headers
 *----------------------------------------------------------------------------*/

/*----------------------------------------------------------------------------
 * Local headers
 *----------------------------------------------------------------------------*/

#include "bft_error.h"
#include "bft_mem.h"
#include "bft_printf.h"

#include "cs_base.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <algorithm>

#include <hipblas.h>
#include <hip/hip_runtime_api.h>

#if (CUDART_VERSION > 10000)
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;
#endif

/*----------------------------------------------------------------------------
 *  Header for the current file
 *----------------------------------------------------------------------------*/

#include "cs_base_accel.h"
#include "cs_parall.h"

#include <hip/hip_cooperative_groups.h>

/*----------------------------------------------------------------------------*/

/*=============================================================================
 * Local Macro Definitions
 *============================================================================*/

#define BLOCKSIZE 256

/*============================================================================
 * Private kernel definitions
 *============================================================================*/

/*----------------------------------------------------------------------------*/

BEGIN_C_DECLS

/*============================================================================
 * Public function definitions
 *============================================================================*/

/*----------------------------------------------------------------------------*/

END_C_DECLS
