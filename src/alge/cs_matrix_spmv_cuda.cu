#include "hip/hip_runtime.h"
/*============================================================================
 * Sparse Matrix-vector multiplication kernels using CUDA.
 *============================================================================*/

/*
  This file is part of code_saturne, a general-purpose CFD tool.

  Copyright (C) 1998-2022 EDF S.A.

  This program is free software; you can redistribute it and/or modify it under
  the terms of the GNU General Public License as published by the Free Software
  Foundation; either version 2 of the License, or (at your option) any later
  version.

  This program is distributed in the hope that it will be useful, but WITHOUT
  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
  FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more
  details.

  You should have received a copy of the GNU General Public License along with
  this program; if not, write to the Free Software Foundation, Inc., 51 Franklin
  Street, Fifth Floor, Boston, MA 02110-1301, USA.
*/

/*----------------------------------------------------------------------------*/

#include "cs_defs.h"

/*----------------------------------------------------------------------------
 * Standard C library headers
 *----------------------------------------------------------------------------*/

#include <stdarg.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <math.h>

#if defined(HAVE_MPI)
#include <mpi.h>
#endif

#if defined(HAVE_CUSPARSE)
#include <hipsparse.h>
#endif

/*----------------------------------------------------------------------------
 * Local headers
 *----------------------------------------------------------------------------*/

#include "bft_mem.h"
#include "bft_error.h"
#include "bft_printf.h"

#include "cs_base.h"
#include "cs_base_cuda.h"
#include "cs_cuda_contrib.h"
#include "cs_halo.h"
#include "cs_halo_perio.h"
#include "cs_log.h"
#include "cs_timer.h"

/*----------------------------------------------------------------------------
 *  Header for the current file
 *----------------------------------------------------------------------------*/

#include "cs_matrix.h"
#include "cs_matrix_priv.h"
#include "cs_matrix_spmv.h"

/*----------------------------------------------------------------------------*/

/*----------------------------------------------------------------------------*/
/*! \file cs_matrix_spmv_cuda.c
 *
 * \brief Sparse Matrix SpMV operations with CUDA.
 */
/*----------------------------------------------------------------------------*/

/*! \cond DOXYGEN_SHOULD_SKIP_THIS */

/*============================================================================
 * Local macro definitions
 *============================================================================*/

#if defined(HAVE_CUSPARSE)
#  if (CUDART_VERSION > 10000)
#    define USE_CUSPARSE_GENERIC_API  1
#  endif
#endif

/*----------------------------------------------------------------------------
 * Compatibility macro for __ldg (load from generic memory) intrinsic,
 * forcing load from read-only texture cache.
 *
 * This was not available in (very old) CUDA architectures.
 *----------------------------------------------------------------------------*/

#if __CUDA_ARCH__ < 350
#define __ldg(ptr) *(ptr);
#endif

/*=============================================================================
 * Local Type Definitions
 *============================================================================*/

#if defined(HAVE_CUSPARSE)

/* Mapping of matrix coefficients and structure to cuSPARSE */
/*----------------------------------------------------------*/

typedef struct _cs_matrix_cusparse_map_t {

#if defined(USE_CUSPARSE_GENERIC_API)

  hipsparseSpMatDescr_t  matA;   /* Handle to cusparse Matrix */
  hipsparseDnVecDescr_t  vecX;   /* Handle to cusparse Vector */
  hipsparseDnVecDescr_t  vecY;   /* Handle to cusparse output Vector */

  void  *vecXValues;  /* Pointer to vector values */
  void  *vecYValues;  /* Pointer to vector values */

  void  *dBuffer;     /* Associated buffer */

#else

  int  nnz;                   /* Number of nonzeroes */
  cusparseMatDescr  *descrA;  /* Handle to cusparse Matrix description */

  void  *d_row_index;         /* Pointer to row index */
  void  *d_col_id;            /* Pointer to column ids */
  void  *d_e_val;             /* Pointer to matrix extradiagonal values */

#endif

} cs_matrix_cusparse_map_t;

#endif // defined(HAVE_CUSPARSE)

/*============================================================================
 *  Global variables
 *============================================================================*/

#if defined(HAVE_CUSPARSE)

static hipsparseHandle_t  _handle = NULL;

#endif

/*============================================================================
 * Private function definitions
 *============================================================================*/

/*----------------------------------------------------------------------------*/
/* \brief Local matrix.vector product y = A.x with CSR matrix arrays.
 *
 * \param[in]   n_rows     number of local rows
 * \param[in]   row_index  pointer to matrix rows index
 * \param[in]   col_id     pointer to matrix column id
 * \param[in]   val        pointer to matrix values
 * \param[in]   x          multipliying vector values
 * \param[out]  y          resulting vector
 */
/*----------------------------------------------------------------------------*/

__global__ static void
_mat_vect_p_l_csr(cs_lnum_t         n_rows,
                  const cs_lnum_t  *__restrict__ row_index,
                  const cs_lnum_t  *__restrict__ col_id,
                  const cs_real_t  *__restrict__ val,
                  const cs_real_t  *__restrict__ x,
                  cs_real_t        *__restrict__ y)
{
  cs_lnum_t ii = blockIdx.x * blockDim.x + threadIdx.x;
  cs_lnum_t jj;

  if (ii < n_rows) {
    cs_real_t sii = 0.0;
    const cs_lnum_t *__restrict__ _col_id = col_id + row_index[ii];
    const cs_real_t *__restrict__ m_row  = val + row_index[ii];
    cs_lnum_t n_cols = row_index[ii + 1] - row_index[ii];
#pragma unroll
    for (jj = 0; jj < n_cols; jj++) {
      sii += m_row[jj] * __ldg(x + _col_id[jj]);
    }
    y[ii] = sii;
  }
}

/*----------------------------------------------------------------------------*/
/* \brief Local matrix.vector product y = A.x with CSR matrix arrays,
 *        excluding diagonal part.
 *
 * \param[in]   n_rows     number of local rows
 * \param[in]   row_index  pointer to matrix rows index
 * \param[in]   col_id     pointer to matrix column id
 * \param[in]   val        pointer to matrix values
 * \param[in]   x          multipliying vector values
 * \param[out]  y          resulting vector
 */
/*----------------------------------------------------------------------------*/

__global__ static void
_mat_vect_p_l_csr_exdiag(cs_lnum_t         n_rows,
                         const cs_lnum_t  *__restrict__ row_index,
                         const cs_lnum_t  *__restrict__ col_id,
                         const cs_real_t  *__restrict__ val,
                         const cs_real_t  *__restrict__ x,
                         cs_real_t        *__restrict__ y)
{
  cs_lnum_t ii = blockIdx.x * blockDim.x + threadIdx.x;

  if (ii < n_rows) {
    cs_real_t        sii            = 0.0;
    const cs_lnum_t *__restrict__ _col_id = col_id + row_index[ii];
    const cs_real_t *__restrict__ m_row  = val + row_index[ii];
    cs_lnum_t n_cols = row_index[ii + 1] - row_index[ii];
#pragma unroll
    for (cs_lnum_t jj = 0; jj < n_cols; jj++) {
      cs_lnum_t c_id = _col_id[jj];
      if (c_id != ii)
        sii += m_row[jj] * __ldg(x + c_id);
    }
    y[ii] = sii;
  }
}

/*----------------------------------------------------------------------------*/
/* \brief Local matrix.vector product y = A.x with MSR matrix arrays.
 *
 * \param[in]   n_rows     number of local rows
 * \param[in]   row_index  pointer to matrix rows index
 * \param[in]   col_id     pointer to matrix column id
 * \param[in]   d_val      pointer to diagonal matrix values
 * \param[in]   x_val      pointer to extradiagonal matrix values
 * \param[in]   x          multipliying vector values
 * \param[out]  y          resulting vector
 */
/*----------------------------------------------------------------------------*/

__global__ static void
_mat_vect_p_l_msr(cs_lnum_t         n_rows,
                  const cs_lnum_t  *__restrict__ row_index,
                  const cs_lnum_t  *__restrict__ col_id,
                  const cs_real_t  *__restrict__ d_val,
                  const cs_real_t  *__restrict__ x_val,
                  const cs_real_t  *__restrict__ x,
                  cs_real_t        *__restrict__ y)
{
  cs_lnum_t ii = blockIdx.x * blockDim.x + threadIdx.x;

  if (ii < n_rows) {
    const cs_lnum_t *__restrict__ _col_id = col_id + row_index[ii];
    const cs_real_t *__restrict__ m_row  = x_val + row_index[ii];

    cs_lnum_t n_cols = row_index[ii + 1] - row_index[ii];

    cs_real_t sii = 0.0;

    for (cs_lnum_t jj = 0; jj < n_cols; jj++)
      sii += m_row[jj] * __ldg(x + _col_id[jj]);

    y[ii] = sii + d_val[ii] * x[ii];
  }
}

/*----------------------------------------------------------------------------*/
/* \brief Local diagonal contribution y = Da.x  + y.
 *
 * This can be combined with a cuSPARSE CSR SpMV product with the
 * extra-diagonal portion of an MSR matrix.
 *
 * \param[in]       n_rows  number of local rows
 * \param[in]       d_val   pointer to diagonal matrix values
 * \param[in]       x       multipliying vector values
 * \param[in, out]  y       resulting vector
 */
/*----------------------------------------------------------------------------*/

__global__ static void
_mat_vect_p_l_msr_adddiag(cs_lnum_t         n_rows,
                          const cs_real_t  *__restrict__ d_val,
                          const cs_real_t  *__restrict__ x,
                          cs_real_t        *__restrict__ y)
{
  cs_lnum_t ii = blockIdx.x * blockDim.x + threadIdx.x;

  if (ii < n_rows)
    y[ii] += d_val[ii] * x[ii];
}

/*----------------------------------------------------------------------------*/
/* \brief Local matrix.vector product y = A.x with MSR matrix,
 *        3x3 blocked diagonal version.
 *
 * \param[in]   n_rows     number of local rows
 * \param[in]   row_index  pointer to matrix rows index
 * \param[in]   col_id     pointer to matrix column id
 * \param[in]   d_val      pointer to diagonal matrix values
 * \param[in]   x_val      pointer to extradiagonal matrix values
 * \param[in]   x          multipliying vector values
 * \param[out]  y          resulting vector
 */
/*----------------------------------------------------------------------------*/

__global__ static void
_b_3_3_mat_vect_p_l_msr(cs_lnum_t        n_rows,
                        const cs_lnum_t  *__restrict__ col_id,
                        const cs_lnum_t  *__restrict__ row_index,
                        const cs_real_t  *__restrict__ x_val,
                        const cs_real_t  *__restrict__ d_val,
                        const cs_real_t  *__restrict__ x,
                        cs_real_t        *__restrict__ y)
{
  cs_lnum_t ii = blockIdx.x * blockDim.x + threadIdx.x;

  if (ii < n_rows) {
    const cs_lnum_t *__restrict__ _col_id = col_id + row_index[ii];
    const cs_real_t *__restrict__ m_row  = x_val + row_index[ii];
    cs_lnum_t n_cols = row_index[ii + 1] - row_index[ii];
    cs_real_t sii[3];
    for (cs_lnum_t kk = 0; kk < 3; kk++) {
      sii[kk] =   d_val[ii * 9 + kk * 3]     * x[ii * 3]
                + d_val[ii * 9 + kk * 3 + 1] * x[ii * 3 + 1]
                + d_val[ii * 9 + kk * 3 + 2] * x[ii * 3 + 2];
    }

    for (cs_lnum_t jj = 0; jj < n_cols; jj++) {
      for (cs_lnum_t kk = 0; kk < 3; kk++)
        sii[kk] += m_row[jj] * __ldg(x + (_col_id[jj]*3 + kk));
    }

    y[ii*3]     = sii[0];
    y[ii*3 + 1] = sii[1];
    y[ii*3 + 2] = sii[2];
  }
}

/*----------------------------------------------------------------------------*/
/* \brief Local matrix.vector product y = A.x with MSR matrix,
 *        excluding 3x3 blocked diagonal.
 *
 * \param[in]   n_rows     number of local rows
 * \param[in]   row_index  pointer to matrix rows index
 * \param[in]   col_id     pointer to matrix column id
 * \param[in]   d_val      pointer to diagonal matrix values
 * \param[in]   x_val      pointer to extradiagonal matrix values
 * \param[in]   x          multipliying vector values
 * \param[out]  y          resulting vector
 */
/*----------------------------------------------------------------------------*/

__global__ static void
_b_3_3_mat_vect_p_l_msr_exdiag(cs_lnum_t        n_rows,
                               const cs_lnum_t  *__restrict__ col_id,
                               const cs_lnum_t  *__restrict__ row_index,
                               const cs_real_t  *__restrict__ x_val,
                               const cs_real_t  *__restrict__ d_val,
                               const cs_real_t  *__restrict__ x,
                               cs_real_t        *__restrict__ y)
{
  cs_lnum_t ii = blockIdx.x * blockDim.x + threadIdx.x;

  if (ii < n_rows) {
    const cs_lnum_t *__restrict__ _col_id = col_id + row_index[ii];
    const cs_real_t *__restrict__ m_row  = x_val + row_index[ii];
    cs_lnum_t n_cols = row_index[ii + 1] - row_index[ii];
    cs_real_t sii[3];
    for (cs_lnum_t kk = 0; kk < 3; kk++)
      sii[kk] = 0.;

    for (cs_lnum_t jj = 0; jj < n_cols; jj++) {
      for (cs_lnum_t kk = 0; kk < 3; kk++)
        sii[kk] += m_row[jj] * __ldg(x + (_col_id[jj]*3 + kk));
    }

    y[ii * 3]     = sii[0];
    y[ii * 3 + 1] = sii[1];
    y[ii * 3 + 2] = sii[2];
  }
}

/*----------------------------------------------------------------------------*/
/* \brief Local matrix.vector product y = A.x with MSR matrix,
 *        blocked diagonal version.
 *
 * \param[in]   n_rows     number of local rows
 * \param[in]   row_index  pointer to matrix rows index
 * \param[in]   col_id     pointer to matrix column id
 * \param[in]   d_val      pointer to diagonal matrix values
 * \param[in]   x_val      pointer to extradiagonal matrix values
 * \param[in]   x          multipliying vector values
 * \param[out]  y          resulting vector
 */
/*----------------------------------------------------------------------------*/

template <const int n>
__global__ static void
_b_mat_vect_p_l_msr(cs_lnum_t        n_rows,
                    const cs_lnum_t  *__restrict__ col_id,
                    const cs_lnum_t  *__restrict__ row_index,
                    const cs_real_t  *__restrict__ x_val,
                    const cs_real_t  *__restrict__ d_val,
                    const cs_real_t  *__restrict__ x,
                    cs_real_t        *__restrict__ y)
{
  cs_lnum_t ii = blockIdx.x * blockDim.x + threadIdx.x;

  if (ii < n_rows) {
    const cs_lnum_t nn = n*n;

    const cs_lnum_t *__restrict__ _col_id = col_id + row_index[ii];
    const cs_real_t *__restrict__ m_row  = x_val + row_index[ii];
    cs_lnum_t n_cols = row_index[ii + 1] - row_index[ii];
    cs_real_t sii[n];

    for (cs_lnum_t kk = 0; kk < n; kk++)
      sii[kk] = 0.;

    for (cs_lnum_t kk = 0; kk < n; kk++) {
      sii[kk] += d_val[ii*nn + kk*n + kk] * x[ii*n + kk];
    }

    for (cs_lnum_t jj = 0; jj < n_cols; jj++) {
      for (cs_lnum_t kk = 0; kk < 3; kk++)
        sii[kk] += m_row[jj] * __ldg(x + (_col_id[jj]*3 + kk));
    }

    for (cs_lnum_t kk = 0; kk < n; kk++)
      y[ii*n + kk] = sii[kk];
  }
}

/*----------------------------------------------------------------------------*/
/* \brief Local matrix.vector product y = A.x with MSR matrix,
 *        excluding blocked diagonal.
 *
 * \param[in]   n_rows     number of local rows
 * \param[in]   row_index  pointer to matrix rows index
 * \param[in]   col_id     pointer to matrix column id
 * \param[in]   d_val      pointer to diagonal matrix values
 * \param[in]   x_val      pointer to extradiagonal matrix values
 * \param[in]   x          multipliying vector values
 * \param[out]  y          resulting vector
 */
/*----------------------------------------------------------------------------*/

template <const int n>
__global__ static void
_b_mat_vect_p_l_msr_exdiag(cs_lnum_t        n_rows,
                           const cs_lnum_t  *__restrict__ col_id,
                           const cs_lnum_t  *__restrict__ row_index,
                           const cs_real_t  *__restrict__ x_val,
                           const cs_real_t  *__restrict__ d_val,
                           const cs_real_t  *__restrict__ x,
                           cs_real_t        *__restrict__ y)
{
  cs_lnum_t ii = blockIdx.x * blockDim.x + threadIdx.x;

  if (ii < n_rows) {
    const cs_lnum_t nn = n*n;

    const cs_lnum_t *__restrict__ _col_id = col_id + row_index[ii];
    const cs_real_t *__restrict__ m_row  = x_val + row_index[ii];
    cs_lnum_t n_cols = row_index[ii + 1] - row_index[ii];
    cs_real_t sii[n];
    for (cs_lnum_t kk = 0; kk < n; kk++)
      sii[kk] = 0.;

    for (cs_lnum_t jj = 0; jj < n_cols; jj++) {
      for (cs_lnum_t kk = 0; kk < n; kk++)
        sii[kk] += m_row[jj] * __ldg(x + (_col_id[jj]*n + kk));
    }

    for (cs_lnum_t kk = 0; kk < n; kk++)
      y[ii*n + kk] = sii[kk];
  }
}

/*----------------------------------------------------------------------------
 * Start synchronization of ghost values prior to matrix.vector product.
 *
 * Values are packed on the device, so:
 * - If MPI is CUDA-aware, no values need to go through the host
 * - Otherwise, only halo values need to go through the host, not the
 *   whole array.
 *
 * parameters:
 *   matrix   <-- pointer to matrix structure
 *   d_x      <-> multipliying vector values (ghost values updated)
 *
 * returns:
 *   halo state to use for synchronisation finalisation.
 *----------------------------------------------------------------------------*/

static cs_halo_state_t *
_pre_vector_multiply_sync_x_start(const cs_matrix_t   *matrix,
                                  cs_real_t            d_x[restrict])
{
 cs_halo_state_t *hs = NULL;

  if (matrix->halo != NULL) {

    hs = cs_halo_state_get_default();

    cs_halo_sync_pack_d(matrix->halo,
                        CS_HALO_STANDARD,
                        CS_REAL_TYPE,
                        matrix->db_size,
                        d_x,
                        NULL,
                        hs);

    cs_halo_sync_start(matrix->halo, d_x, hs);

  }

  return hs;
}

/*----------------------------------------------------------------------------
 * Unset matrix cuSPARSE mapping.
 *
 * parameters:
 *   matrix    <-- pointer to matrix structure
 *----------------------------------------------------------------------------*/

static void
_unset_cusparse_map(cs_matrix_t   *matrix)
{
  cs_matrix_cusparse_map_t *csm
    = (cs_matrix_cusparse_map_t *)matrix->ext_lib_map;

  if (csm == NULL)
    return;

#if defined(USE_CUSPARSE_GENERIC_API)

  hipsparseDestroySpMat(csm->matA);

  if (csm->vecXValues != NULL)
    hipsparseDestroyDnVec(csm->vecX);
  if (csm->vecYValues != NULL)
    hipsparseDestroyDnVec(csm->vecY);

  if (csm->dBuffer != NULL) {
    CS_CUDA_CHECK(hipFree(csm->dBuffer));
    csm->dBuffer = NULL;
  }

#else

  hipsparseDestroyMatDescr(csm->descrA);

#endif

  BFT_FREE(matrix->ext_lib_map);
}

/*----------------------------------------------------------------------------
 * Set matrix cuSPARSE mapping.
 *
 * parameters:
 *   matrix    <-- pointer to matrix structure
 *----------------------------------------------------------------------------*/

static cs_matrix_cusparse_map_t *
_set_cusparse_map(cs_matrix_t   *matrix)
{
  cs_matrix_cusparse_map_t *csm
    = (cs_matrix_cusparse_map_t *)matrix->ext_lib_map;

  if (csm != NULL) {
    _unset_cusparse_map(matrix);
  }
  else {
    BFT_MALLOC(csm, 1, cs_matrix_cusparse_map_t);
    matrix->ext_lib_map = (void *)csm;
  }
  matrix->destroy_adaptor = _unset_cusparse_map;

  void *row_index, *col_id;
  void *e_val;
  cs_lnum_t nnz = 0;

  if (matrix->type == CS_MATRIX_CSR) {
    const cs_matrix_struct_csr_t *ms
      = (const cs_matrix_struct_csr_t  *)matrix->structure;
    const cs_matrix_coeff_csr_t *mc
      = (const cs_matrix_coeff_csr_t *)matrix->coeffs;
    nnz = ms->row_index[matrix->n_rows];
    row_index = cs_get_device_ptr(const_cast<cs_lnum_t *>(ms->row_index));
    col_id = cs_get_device_ptr(const_cast<cs_lnum_t *>(ms->col_id));
    e_val = cs_get_device_ptr(const_cast<cs_real_t *>(mc->val));
  }
  else {
    const cs_matrix_struct_dist_t *ms
      = (const cs_matrix_struct_dist_t *)matrix->structure;
    const cs_matrix_coeff_dist_t *mc
      = (const cs_matrix_coeff_dist_t *)matrix->coeffs;
    nnz = ms->e.row_index[matrix->n_rows];
    row_index = cs_get_device_ptr(const_cast<cs_lnum_t *>(ms->e.row_index));
    col_id = cs_get_device_ptr(const_cast<cs_lnum_t *>(ms->e.col_id));
    e_val = cs_get_device_ptr(const_cast<cs_real_t *>(mc->e_val));
  }

  hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS;

  if (_handle == NULL)
    status = hipsparseCreate(&_handle);

#if defined(USE_CUSPARSE_GENERIC_API)

  if (HIPSPARSE_STATUS_SUCCESS != status)
    bft_error(__FILE__, __LINE__, 0, _("%s: %s."),
              __func__, hipsparseGetErrorString(status));

  hipsparseIndexType_t index_dtype
    = (sizeof(cs_lnum_t) == 4) ? HIPSPARSE_INDEX_32I : HIPSPARSE_INDEX_64I;
  hipDataType val_dtype
    = (sizeof(cs_real_t) == 8) ? HIP_R_64F : HIP_R_32F;

  csm->vecXValues = NULL;  /* Pointer to vector values */
  csm->vecYValues = NULL;  /* Pointer to vector values */
  csm->dBuffer = NULL;

  status = hipsparseCreateCsr(&(csm->matA),
                             matrix->n_rows,
                             matrix->n_cols_ext,
                             nnz,
                             row_index,
                             col_id,
                             e_val,
                             index_dtype,
                             index_dtype,
                             HIPSPARSE_INDEX_BASE_ZERO,
                             val_dtype);

  if (HIPSPARSE_STATUS_SUCCESS != status)
    bft_error(__FILE__, __LINE__, 0, _("%s: %s."),
              __func__, hipsparseGetErrorString(status));

#else

  if (HIPSPARSE_STATUS_SUCCESS != status)
    bft_error(__FILE__, __LINE__, 0, _("%s: cuSPARSE error %d."),
              __func__, (int)status);

  csm->nnz = nnz;
  csm->d_e_val = e_val;

  csm->d_row_index = row_index;
  csm->d_col_id = col_id;
  csm->d_e_val = e_val;

  status = hipsparseCreateMatDescr(&(csm->descrA));

  if (HIPSPARSE_STATUS_SUCCESS != status)
    bft_error(__FILE__, __LINE__, 0, _("%s: cuSPARSE error %d."),
              __func__, (int)status);

  hipsparseSetMatIndexBase(csm->descrA, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseSetMatType(csm->descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatDiagType(csm->descrA, HIPSPARSE_DIAG_TYPE_NON_UNIT);

#endif

  return csm;
}

/*----------------------------------------------------------------------------
 * Update matrix cuSPARSE mapping.
 *
 * parameters:
 *   csm       <-> cuSPARSE matrix mapping
 *   matrix    <-- pointer to matrix structure
 *   d_x       <-- pointer to input vector (on device)
 *   d_y       <-- pointer to output vector (on device)
 *----------------------------------------------------------------------------*/

static void
_update_cusparse_map(cs_matrix_cusparse_map_t  *csm,
                     const cs_matrix_t         *matrix,
                     void                      *d_x,
                     void                      *d_y)
{
  assert(csm != NULL);

#if defined(USE_CUSPARSE_GENERIC_API)

  hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS;
  hipDataType val_dtype
    = (sizeof(cs_real_t) == 8) ? HIP_R_64F : HIP_R_32F;

  if (d_x != csm->vecXValues) {
    if (csm->vecXValues != NULL)
      hipsparseDestroyDnVec(csm->vecX);

    status = hipsparseCreateDnVec(&(csm->vecX),
                                 matrix->n_cols_ext,
                                 d_x,
                                 val_dtype);

    if (HIPSPARSE_STATUS_SUCCESS != status)
      bft_error(__FILE__, __LINE__, 0, _("%s: %s."),
                __func__, hipsparseGetErrorString(status));

    csm->vecXValues = d_x;
  }

  if (d_y != csm->vecYValues) {
    if (csm->vecYValues != NULL)
      hipsparseDestroyDnVec(csm->vecY);

    status = hipsparseCreateDnVec(&(csm->vecY),
                                 matrix->n_rows,
                                 d_y,
                                 val_dtype);

    if (HIPSPARSE_STATUS_SUCCESS != status)
      bft_error(__FILE__, __LINE__, 0, _("%s: %s."),
                __func__, hipsparseGetErrorString(status));

    csm->vecYValues = d_y;
  }

  if (csm->dBuffer == NULL) {
    size_t bufferSize = 0;
    cs_real_t alpha = 1.0;
    cs_real_t beta = 1.0;  /* 0 should be enough for SmPV, 1 needed for
                              y = A.x + b.y
                              which is useful when y is initialized by
                              a separate diagonal da.x product */

    status = hipsparseSpMV_bufferSize(_handle,
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     &alpha,
                                     csm->matA,
                                     csm->vecX,
                                     &beta,
                                     csm->vecY,
                                     val_dtype,
                                     HIPSPARSE_MV_ALG_DEFAULT,
                                     &bufferSize);

    CS_CUDA_CHECK(hipMalloc(&(csm->dBuffer), bufferSize));
  }

#endif
}

/*! (DOXYGEN_SHOULD_SKIP_THIS) \endcond */

BEGIN_C_DECLS

/*============================================================================
 * Public function definitions
 *============================================================================*/

/*----------------------------------------------------------------------------*/
/*!
 * \brief Finalize CUDA matrix API.
 *
 * This frees resources such as the cuSPARSE handle, if used.
 */
/*----------------------------------------------------------------------------*/

void
cs_matrix_spmv_cuda_finalize(void)
{
  if (_handle != NULL) {
    hipsparseDestroy(_handle);
    _handle = NULL;
  }
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Matrix.vector product y = A.x with CSR matrix, scalar CUDA version.
 *
 * \param[in]   matrix        pointer to matrix structure
 * \param[in]   exclude_diag  exclude diagonal if true,
 * \param[in]   sync          synchronize ghost cells if true
 * \param[in]   x             multipliying vector values
 * \param[out]  y             resulting vector
 */
/*----------------------------------------------------------------------------*/

void
cs_matrix_spmv_cuda_p_l_csr(const cs_matrix_t  *matrix,
                            bool                exclude_diag,
                            bool                sync,
                            cs_real_t           x[restrict],
                            cs_real_t           y[restrict])
{
  const cs_matrix_struct_csr_t *ms
    = (const cs_matrix_struct_csr_t *)matrix->structure;
  const cs_matrix_coeff_csr_t *mc
    = (const cs_matrix_coeff_csr_t  *)matrix->coeffs;

  const cs_lnum_t *__restrict__ d_col_id
    = (const cs_lnum_t *)cs_get_device_ptr(const_cast<cs_lnum_t *>(ms->col_id));
  const cs_lnum_t *__restrict__ d_row_index
    = (const cs_lnum_t *)cs_get_device_ptr
                           (const_cast<cs_lnum_t *>(ms->row_index));
  const cs_real_t *__restrict__ d_val
    = (const cs_real_t *)cs_get_device_ptr(const_cast<cs_real_t *>(mc->val));

  cs_real_t *__restrict__ d_x
    = (cs_real_t *)cs_get_device_ptr(const_cast<cs_real_t *>(x));
  cs_real_t *__restrict__  d_y
    = (cs_real_t *)cs_get_device_ptr(const_cast<cs_real_t *>(y));

  /* Ghost cell communication */

  if (sync) {
    cs_halo_state_t *hs = _pre_vector_multiply_sync_x_start(matrix, d_x);
    cs_halo_sync_wait(matrix->halo, d_x, hs);
  }

  /* Compute SpMV */

  unsigned int blocksize = 256;
  unsigned int gridsize
    = (unsigned int)ceil((double)ms->n_rows / blocksize);

  if (!exclude_diag)
    _mat_vect_p_l_csr<<<gridsize, blocksize>>>
      (ms->n_rows, d_col_id, d_row_index, d_val, d_x, d_y);
  else
    _mat_vect_p_l_csr_exdiag<<<gridsize, blocksize>>>
      (ms->n_rows, d_col_id, d_row_index, d_val, d_x, d_y);

  // hipDeviceSynchronize();
  // CS_CUDA_CHECK(hipGetLastError());
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Matrix.vector product y = A.x with CSR matrix, scalar cuSPARSE version.
 *
 * \param[in]   matrix        pointer to matrix structure
 * \param[in]   exclude_diag  exclude diagonal if true,
 * \param[in]   sync          synchronize ghost cells if true
 * \param[in]   x             multipliying vector values
 * \param[out]  y             resulting vector
 */
/*----------------------------------------------------------------------------*/

void
cs_matrix_spmv_cuda_p_l_csr_cusparse(cs_matrix_t  *matrix,
                                     bool          exclude_diag,
                                     bool          sync,
                                     cs_real_t     x[restrict],
                                     cs_real_t     y[restrict])
{
  cs_matrix_cusparse_map_t *csm
    = (cs_matrix_cusparse_map_t *)matrix->ext_lib_map;

  void  *d_x = cs_get_device_ptr(const_cast<cs_real_t *>(x));
  void  *d_y = cs_get_device_ptr(y);

  if (csm == NULL) {
    matrix->ext_lib_map = _set_cusparse_map(matrix);
    csm = (cs_matrix_cusparse_map_t *)matrix->ext_lib_map;
  }

  /* Ghost cell communication */

  if (sync) {
    cs_halo_state_t *hs = _pre_vector_multiply_sync_x_start(matrix,
                                                            (cs_real_t *)d_x);
    cs_halo_sync_wait(matrix->halo, (cs_real_t *)d_x, hs);
  }

  _update_cusparse_map(csm, matrix, d_x, d_y);

  cs_real_t alpha = 1.0;
  cs_real_t beta = 0.0;

#if defined(USE_CUSPARSE_GENERIC_API)

  hipDataType val_dtype
    = (sizeof(cs_real_t) == 8) ? HIP_R_64F : HIP_R_32F;

  hipsparseStatus_t status = hipsparseSpMV(_handle,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         &alpha,
                                         csm->matA,
                                         csm->vecX,
                                         &beta,
                                         csm->vecY,
                                         val_dtype,
                                         HIPSPARSE_MV_ALG_DEFAULT,
                                         csm->dBuffer);

#else

#if SIZEOF_DOUBLE == 8

    hipsparseDcsrmv(_handle,
                   HIPSPARSE_OPERATION_NON_TRANSPOSE,
                   matrix->n_rows,
                   matrix->n_cols_ext,
                   csm->nnz,
                   &alpha,
                   csm->descrA,
                   (const double *)csm->d_e_val,
                   (const int *)csm->d_row_index,
                   (const int *)csm->d_col_id,
                   (const double *)d_x,
                   &beta,
                   (double *)d_y);

#elif SIZEOF_DOUBLE == 4

    hipsparseScsrmv(_handle,
                   HIPSPARSE_OPERATION_NON_TRANSPOSE,
                   matrix->n_rows,
                   matrix->n_cols_ext,
                   csm->nnz,
                   &alpha,
                   csm->descrA,
                   (const float *)csm->d_e_val,
                   (const int *)csm->d_row_index,
                   (const int *)csm->d_col_id,
                   (const float *)d_x,
                   &beta,
                   (float *)d_y);

#endif

#endif

  hipDeviceSynchronize();
  CS_CUDA_CHECK(hipGetLastError());
}

/*----------------------------------------------------------------------------*/

END_C_DECLS
