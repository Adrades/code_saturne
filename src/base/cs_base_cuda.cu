#include "hip/hip_runtime.h"
/*============================================================================
 * Low-level functions and global variables definition for CUDA.
 *============================================================================*/

/*
  This file is part of Code_Saturne, a general-purpose CFD tool.

  Copyright (C) 1998-2021 EDF S.A.

  This program is free software; you can redistribute it and/or modify it under
  the terms of the GNU General Public License as published by the Free Software
  Foundation; either version 2 of the License, or (at your option) any later
  version.

  This program is distributed in the hope that it will be useful, but WITHOUT
  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
  FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more
  details.

  You should have received a copy of the GNU General Public License along with
  this program; if not, write to the Free Software Foundation, Inc., 51 Franklin
  Street, Fifth Floor, Boston, MA 02110-1301, USA.
*/

/*----------------------------------------------------------------------------*/

#include "cs_defs.h"

/*----------------------------------------------------------------------------
 * Standard C library headers
 *----------------------------------------------------------------------------*/

/*----------------------------------------------------------------------------
 * Local headers
 *----------------------------------------------------------------------------*/

#include "bft_error.h"
#include "bft_mem.h"
#include "bft_printf.h"

#include "cs_base.h"
#include "cs_log.h"

/*----------------------------------------------------------------------------
 *  Header for the current file
 *----------------------------------------------------------------------------*/

#include "cs_base_cuda.h"

/*----------------------------------------------------------------------------*/

BEGIN_C_DECLS

/*! \cond DOXYGEN_SHOULD_SKIP_THIS */

/*============================================================================
 * Local Macro Definitions
 *============================================================================*/

#define CS_CUDA_CHECK(x)                                                       \
if (hipError_t err = (x)) {                                                   \
  bft_error(__FILE__, __LINE__, 0, _("CUDA error: %s"), hipGetErrorString(err)); \
}

/*============================================================================
 * Local Type Definitions
 *============================================================================*/

/*============================================================================
 *  Global variables
 *============================================================================*/

int  cs_glob_cuda_device_id = -1;

/*============================================================================
 * Private function definitions
 *============================================================================*/

/*============================================================================
 * Semi-private function prototypes
 *
 * The following functions are intended to be used by the common
 * host-device memory management functions from cs_base_accel.c, and
 * not directly by the user.
 *============================================================================*/

/*----------------------------------------------------------------------------*/
/*!
 * \brief Allocate n bytes of CUDA device memory.
 *
 * This function simply wraps hipMallocManaged, which could probably be
 * directly called from C or C++, but whose use in such manner is not
 * well documented, and whose declaration in hip/hip_runtime.h requires
 * support of function attributes by compiler.
 *
 * A safety check is added.
 *
 * \param [in]  n          element size
 * \param [in]  var_name   allocated variable name string
 * \param [in]  file_name  name of calling source file
 * \param [in]  line_num   line number in calling source file
 *
 * \returns pointer to allocated memory.
 */
/*----------------------------------------------------------------------------*/

void *
cs_cuda_mem_malloc_device(size_t        n,
                          const char   *var_name,
                          const char   *file_name,
                          int           line_num)
{
  void *ptr = NULL;

  CS_CUDA_CHECK_CALL(hipMalloc(&ptr, n), file_name, line_num);

  return ptr;
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Allocate n bytes of CUDA managed memory.
 *
 * This function simply wraps hipMallocManaged, which could probably be
 * directly called from C or C++, but whose use in such manner is not
 * well documented, and whose declaration in hip/hip_runtime.h requires
 * support of function attributes by compiler.
 *
 * A safety check is added.
 *
 * \param [in]  n          element size
 * \param [in]  var_name   allocated variable name string
 * \param [in]  file_name  name of calling source file
 * \param [in]  line_num   line number in calling source file
 *
 * \returns pointer to allocated memory.
 */
/*----------------------------------------------------------------------------*/

void *
cs_cuda_mem_malloc_managed(size_t        n,
                           const char   *var_name,
                           const char   *file_name,
                           int           line_num)
{
  void *ptr = NULL;

  CS_CUDA_CHECK_CALL(hipMallocManaged(&ptr, n), file_name, line_num);

#if 0
  CS_CUDA_CHECK_CALL(hipMemPrefetchAsync (*pointer, size, hipCpuDeviceId, 0),
                     file_name, line_num);
  CS_CUDA_CHECK_CALL(hipDeviceSynchronize(), file_name, line_num);
#endif

  return ptr;
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Free CUDA memory associated with a given pointer.
 *
 * This function simply wraps hipFree, which could probably be
 * directly called from C or C++, but whose use in such manner is not
 * well documented, and whose declaration in hip/hip_runtime.h requires
 * support of function attributes by compiler.
 *
 * A safety check is added.
 *
 * \param [in]  p          pointer to device memory
 * \param [in]  var_name   allocated variable name string
 * \param [in]  file_name  name of calling source file
 * \param [in]  line_num   line number in calling source file
 *
 * \returns pointer to allocated memory.
 */
/*----------------------------------------------------------------------------*/

void
cs_cuda_mem_free(void         *p,
                 const char   *var_name,
                 const char   *file_name,
                 int           line_num)
{
  CS_CUDA_CHECK_CALL(hipFree(p), file_name, line_num);

#if 0
  CS_CUDA_CHECK_CALL((hipDeviceSynchronize(), file_name, line_num);
#endif
}

/*! (DOXYGEN_SHOULD_SKIP_THIS) \endcond */

/*============================================================================
 * Public function definitions
 *============================================================================*/

/*----------------------------------------------------------------------------*/
/*!
 * \brief  Log information on available CUDA devices.
 *
 * \param[in]  log_id  id of log file in which to print information
 */
/*----------------------------------------------------------------------------*/

void
cs_base_cuda_device_info(cs_log_t  log_id)
{
  int n_devices = 0;

  hipError_t retval = hipGetDeviceCount(&n_devices);

  if (retval == hipErrorNoDevice)
    cs_log_printf(log_id,
                  _("  CUDA device:         none available\n"));
  else if (retval)
    cs_log_printf(log_id,
                  _("  CUDA device:         %s\n"),
		  hipGetErrorString(retval));

  char buffer[256] = "";

  for (int i = 0; i < n_devices; i++) {
    struct hipDeviceProp_t prop;
    CS_CUDA_CHECK(hipGetDeviceProperties(&prop, i));
    unsigned long long mem = prop.totalGlobalMem / 1000000;
    char mode_name[32] = "";
    if (prop.computeMode == hipComputeModeDefault)
      snprintf(mode_name, 31, "default");
    else if (prop.computeMode == hipComputeModeExclusive)
      snprintf(mode_name, 31, "exclusive");
    else if (prop.computeMode == hipComputeModeProhibited)
      snprintf(mode_name, 31, "prohibited");

    cs_log_printf
      (log_id,
       _("  CUDA device %d:       %s\n"),
       i, prop.name);

    if (strncmp(prop.name, buffer, 255) != 0)
      cs_log_printf
        (log_id,
         _("                       Compute capability: %d.%d\n"
           "                       Memory: %llu %s\n"
           "                       Integrated: %d\n"
           "                       Can map host memory: %d\n"
           "                       Compute mode: %s\n"),
         prop.major, prop.minor,
         mem, _("MB"),
         prop.integrated,
         prop.canMapHostMemory, mode_name);

    strncpy(buffer, prop.name, 255);
    buffer[255] = '\0';
  }
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Set CUDA device based on MPI rank and number of devices.
 *
 * \param[in]  comm            associated MPI communicator
 * \param[in]  ranks_per_node  number of ranks per node (min and max)
 */
/*----------------------------------------------------------------------------*/

void
cs_base_cuda_set_default_device(void)
{
  int device_id = 0, n_devices = 0;

  hipError_t ret_code = hipGetDeviceCount(&n_devices);

  if (ret_code == hipErrorNoDevice)
    return;

  if (hipSuccess != ret_code)
    bft_error(__FILE__, __LINE__, 0, "[CUDA errror] %d: %s\n  running: %s",
              ret_code, ::hipGetErrorString(ret_code), __func__);

#if defined(HAVE_MPI)

  if (cs_glob_rank_id > -1 && n_devices > 1) {

    MPI_Comm  comm = cs_glob_mpi_comm;
    int       max_ranks_per_node = -1;

    /* get local rank */

    MPI_Comm sh_comm;
    MPI_Comm_split_type(comm, MPI_COMM_TYPE_SHARED, 0,
                        MPI_INFO_NULL, &sh_comm);
    int sh_rank;
    MPI_Comm_rank(sh_comm, &sh_rank);

    MPI_Allreduce(MPI_IN_PLACE, &sh_rank, 1, MPI_INT, MPI_MAX, sh_comm);
    MPI_Comm_free(&sh_comm);

    MPI_Allreduce(&sh_rank, &max_ranks_per_node, 1, MPI_INT, MPI_MAX, comm);
    max_ranks_per_node += 1;

    int n_ranks_per_device = max_ranks_per_node / n_devices;

    device_id = sh_rank / n_ranks_per_device;

  }

#endif

  CS_CUDA_CHECK_CALL(hipSetDevice(device_id), __FILE__, __LINE__);
}

/*----------------------------------------------------------------------------*/

END_C_DECLS
